#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>

typedef struct {
    int *data;
    int size;
    int cap;
} IntVec;

void vec_init(IntVec *v) {
    v->cap = 4;
    v->size = 0;
    v->data = (int*)malloc(v->cap * sizeof(int));
}

void vec_push(IntVec *v, int x) {
    if (v->size == v->cap) {
        v->cap *= 2;
        v->data = (int*)realloc(v->data, v->cap * sizeof(int));
    }
    v->data[v->size++] = x;
}

void vec_free(IntVec *v) {
    free(v->data);
    v->data = NULL;
    v->size = v->cap = 0;
}

// CUDA kernel to initialize random number generators
__global__ void init_curand(hiprandState *state, unsigned long seed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

// CUDA kernel to generate candidate edges
__global__ void generate_edges(int *edges, int n, int m, hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < m) {
        hiprandState local_state = state[idx];
        int u = hiprand(&local_state) % n;
        int v = hiprand(&local_state) % n;
        edges[idx * 2] = u;
        edges[idx * 2 + 1] = v;
        state[idx] = local_state;
    }
}

// CUDA kernel to validate edges (no self-loops, ensure u < v for uniqueness)
__global__ void validate_edges(int *edges, int *valid, int m, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < m) {
        int u = edges[idx * 2];
        int v = edges[idx * 2 + 1];
        valid[idx] = (u != v && u >= 0 && u < n && v >= 0 && v < n && u < v) ? 1 : 0;
    }
}

// CUDA kernel to compute degrees for CSR row_ptr
__global__ void compute_degrees(int *row_ptr, int *degrees, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        degrees[idx] = row_ptr[idx + 1] - row_ptr[idx];
    }
}

// CUDA kernel to find min-degree unvisited node
__global__ void find_min_degree_unvisited(int *degrees, int *visited, int n, int *min_deg, int *min_idx) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n && !visited[idx]) {
        int deg = degrees[idx];
        atomicMin(min_deg, deg);
        if (deg == *min_deg) {
            atomicMin(min_idx, idx);
        }
    }
}

// CUDA kernel to mark visited nodes and build queue
__global__ void mark_visited(int *col_idx, int start, int size, int *visited, int *queue, int *qtail) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int v = col_idx[start + idx];
        if (!visited[v]) {
            visited[v] = 1;
            int pos = atomicAdd(qtail, 1);
            queue[pos] = v;
        }
    }
}

void generate_random_graph_gpu(int n, int m, int **d_row_ptr, int **d_col_idx, int *total_edges) {
    int block_size = 256;
    int grid_size = (m + block_size - 1) / block_size;

    // Allocate GPU memory for edge generation
    int *d_edges, *d_valid;
    hiprandState *d_state;
    hipMalloc(&d_edges, m * 2 * sizeof(int));
    hipMalloc(&d_valid, m * sizeof(int));
    hipMalloc(&d_state, m * sizeof(hiprandState));

    // Initialize random number generators
    init_curand<<<grid_size, block_size>>>(d_state, time(NULL), m);
    hipDeviceSynchronize();

    // Generate candidate edges
    generate_edges<<<grid_size, block_size>>>(d_edges, n, m, d_state);
    hipDeviceSynchronize();

    // Validate edges
    validate_edges<<<grid_size, block_size>>>(d_edges, d_valid, m, n);
    hipDeviceSynchronize();

    // Count valid edges using CUB
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    int *d_valid_count;
    hipMalloc(&d_valid_count, sizeof(int));
    hipMemset(d_valid_count, 0, sizeof(int));
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_valid, d_valid_count, m);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_valid, d_valid_count, m);
    int valid_edges;
    hipMemcpy(&valid_edges, d_valid_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Allocate memory for CSR format
    hipMalloc(d_row_ptr, (n + 1) * sizeof(int));
    hipMemset(*d_row_ptr, 0, (n + 1) * sizeof(int));
    hipMalloc(d_col_idx, valid_edges * 2 * sizeof(int));

    // Build CSR format (row_ptr and col_idx)
    int *d_edge_counts;
    hipMalloc(&d_edge_counts, n * sizeof(int));
    hipMemset(d_edge_counts, 0, n * sizeof(int));

    // Count edges per node
    grid_size = (m + block_size - 1) / block_size;
    for (int i = 0; i < m; ++i) {
        int u, v, valid;
        hipMemcpy(&u, d_edges + i * 2, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&v, d_edges + i * 2 + 1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&valid, d_valid + i, sizeof(int), hipMemcpyDeviceToHost);
        if (valid) {
            int one = 1;
            hipMemcpy(d_edge_counts + u, &one, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_edge_counts + v, &one, sizeof(int), hipMemcpyHostToDevice);
        }
    }

    // Compute row_ptr using CUB prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_edge_counts, *d_row_ptr + 1, n);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_edge_counts, *d_row_ptr + 1, n);
    hipMemcpy(total_edges, *d_row_ptr + n, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Populate col_idx
    int *h_col_idx = (int*)malloc(valid_edges * 2 * sizeof(int));
    int pos = 0;
    for (int i = 0; i < m; ++i) {
        int u, v, valid;
        hipMemcpy(&u, d_edges + i * 2, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&v, d_edges + i * 2 + 1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&valid, d_valid + i, sizeof(int), hipMemcpyDeviceToHost);
        if (valid) {
            h_col_idx[pos++] = v; // u -> v
            h_col_idx[pos++] = u; // v -> u (undirected)
        }
    }
    hipMemcpy(*d_col_idx, h_col_idx, valid_edges * 2 * sizeof(int), hipMemcpyHostToDevice);

    // Cleanup
    hipFree(d_edges);
    hipFree(d_valid);
    hipFree(d_state);
    hipFree(d_valid_count);
    hipFree(d_edge_counts);
    hipFree(d_temp_storage);
    free(h_col_idx);
}

int *rcm_cuda(int *d_row_ptr, int *d_col_idx, int n, int total_edges) {
    // Allocate GPU memory
    int *d_degrees, *d_visited, *d_queue, *d_order;
    int *d_min_deg, *d_min_idx, *d_qtail, *d_visited_count;
    hipMalloc(&d_degrees, n * sizeof(int));
    hipMalloc(&d_visited, n * sizeof(int));
    hipMalloc(&d_queue, n * sizeof(int));
    hipMalloc(&d_order, n * sizeof(int));
    hipMalloc(&d_min_deg, sizeof(int));
    hipMalloc(&d_min_idx, sizeof(int));
    hipMalloc(&d_qtail, sizeof(int));
    hipMalloc(&d_visited_count, sizeof(int));

    hipMemset(d_visited, 0, n * sizeof(int));
    hipMemset(d_visited_count, 0, sizeof(int));

    // Compute degrees
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    compute_degrees<<<grid_size, block_size>>>(d_row_ptr, d_degrees, n);
    hipDeviceSynchronize();

    int *order = (int*)malloc(n * sizeof(int));
    int order_pos = 0;
    int visited_count = 0;

    while (visited_count < n) {
        int h_min_deg = INT_MAX, h_min_idx = -1;
        hipMemset(d_min_deg, 0x7f, sizeof(int));
        hipMemset(d_min_idx, 0x7f, sizeof(int));
        find_min_degree_unvisited<<<grid_size, block_size>>>(d_degrees, d_visited, n, d_min_deg, d_min_idx);
        hipMemcpy(&h_min_deg, d_min_deg, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_min_idx, d_min_idx, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if (h_min_idx == -1) break;

        int qhead = 0, qtail = 0;
        int *h_queue = (int*)malloc(n * sizeof(int));
        h_queue[qtail++] = h_min_idx;
        int one = 1;
        hipMemcpy(d_visited + h_min_idx, &one, sizeof(int), hipMemcpyHostToDevice);
        visited_count++;

        while (qhead < qtail) {
            int u = h_queue[qhead++];
            order[order_pos++] = u;

            // Sort neighbors by degree
            int start = 0;
            hipMemcpy(&start, d_row_ptr + u, sizeof(int), hipMemcpyDeviceToHost);
            int size = 0;
            hipMemcpy(&size, d_row_ptr + u + 1, sizeof(int), hipMemcpyDeviceToHost);
            size -= start;
            if (size > 0) {
                void *d_temp_storage = NULL;
                size_t temp_storage_bytes = 0;
                int *d_keys = d_col_idx + start;
                int *d_values;
                hipMalloc(&d_values, size * sizeof(int));

                // Map degrees to neighbors
                for (int i = 0; i < size; ++i) {
                    int v;
                    hipMemcpy(&v, d_col_idx + start + i, sizeof(int), hipMemcpyDeviceToHost);
                    int deg;
                    hipMemcpy(&deg, d_degrees + v, sizeof(int), hipMemcpyDeviceToHost);
                    hipMemcpy(d_values + i, &deg, sizeof(int), hipMemcpyHostToDevice);
                }

                hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                                d_values, d_values, d_keys, d_keys, size);
                hipMalloc(&d_temp_storage, temp_storage_bytes);
                hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                                d_values, d_values, d_keys, d_keys, size);
                hipFree(d_temp_storage);
                hipFree(d_values);
            }

            // Mark unvisited neighbors
            hipMemset(d_qtail, 0, sizeof(int));
            mark_visited<<<(size + block_size - 1) / block_size, block_size>>>(d_col_idx, start, size, d_visited, d_queue, d_qtail);
            hipMemcpy(&qtail, d_qtail, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_queue + qhead, d_queue, qtail * sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            visited_count += qtail;
        }
        free(h_queue);
    }

    // Create permutation
    int *perm = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < n; ++i) perm[i] = order[n - 1 - i];

    // Cleanup
    hipFree(d_degrees);
    hipFree(d_visited);
    hipFree(d_queue);
    hipFree(d_order);
    hipFree(d_min_deg);
    hipFree(d_min_idx);
    hipFree(d_qtail);
    hipFree(d_visited_count);
    free(order);

    return perm;
}

int main(int argc, char **argv) {
    int n, m;
    int *d_row_ptr, *d_col_idx;
    int total_edges = 0;

    if (argc == 4 && strcmp(argv[1], "--random") == 0) {
        n = atoi(argv[2]);
        m = atoi(argv[3]);
        if (n <= 0 || m < 0) {
            fprintf(stderr, "Invalid N or M\n");
            return 1;
        }
        generate_random_graph_gpu(n, m, &d_row_ptr, &d_col_idx, &total_edges);

        // Print generated graph for verification
        // int *h_row_ptr = (int*)malloc((n + 1) * sizeof(int));
        // int *h_col_idx = (int*)malloc(total_edges * sizeof(int));
        // hipMemcpy(h_row_ptr, d_row_ptr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(h_col_idx, d_col_idx, total_edges * sizeof(int), hipMemcpyDeviceToHost);
        // printf("# Generated random graph:\n%d %d\n", n, total_edges / 2);
        // for (int i = 0; i < n; ++i) {
        //     for (int j = h_row_ptr[i]; j < h_row_ptr[i + 1]; ++j) {
        //         int v = h_col_idx[j];
        //         if (i < v) { // Print each edge once
        //             printf("%d %d\n", i, v);
        //         }
        //     }
        // }
        // printf("# End of graph\n");
        // free(h_row_ptr);
        // free(h_col_idx);
    } else {
        if (scanf("%d %d", &n, &m) != 2) {
            fprintf(stderr, "Expected: n m\n");
            return 1;
        }
        // Read edges on CPU
        int *h_edges = (int*)malloc(m * 2 * sizeof(int));
        int valid_edges = 0;
        for (int i = 0; i < m; ++i) {
            int u, v;
            if (scanf("%d %d", &u, &v) != 2) {
                fprintf(stderr, "Expected edge list\n");
                free(h_edges);
                return 1;
            }
            if (u < 0 || u >= n || v < 0 || v >= n || u == v) continue;
            h_edges[valid_edges * 2] = u < v ? u : v;
            h_edges[valid_edges * 2 + 1] = u < v ? v : u;
            valid_edges++;
        }

        // Build CSR on GPU
        int *d_edge_counts;
        hipMalloc(&d_edge_counts, n * sizeof(int));
        hipMemset(d_edge_counts, 0, n * sizeof(int));
        total_edges = valid_edges * 2;
        hipMalloc(&d_row_ptr, (n + 1) * sizeof(int));
        hipMemset(d_row_ptr, 0, (n + 1) * sizeof(int));
        hipMalloc(&d_col_idx, total_edges * sizeof(int));

        // Count edges per node
        for (int i = 0; i < valid_edges; ++i) {
            int u = h_edges[i * 2];
            int v = h_edges[i * 2 + 1];
            int one = 1;
            hipMemcpy(d_edge_counts + u, &one, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_edge_counts + v, &one, sizeof(int), hipMemcpyHostToDevice);
        }

        // Compute row_ptr
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_edge_counts, d_row_ptr + 1, n);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_edge_counts, d_row_ptr + 1, n);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        // Populate col_idx
        int *h_col_idx = (int*)malloc(total_edges * sizeof(int));
        int pos = 0;
        for (int i = 0; i < valid_edges; ++i) {
            h_col_idx[pos++] = h_edges[i * 2 + 1]; // u -> v
            h_col_idx[pos++] = h_edges[i * 2];     // v -> u
        }
        hipMemcpy(d_col_idx, h_col_idx, total_edges * sizeof(int), hipMemcpyHostToDevice);
        free(h_edges);
        free(h_col_idx);
        hipFree(d_edge_counts);
    }

    int *perm = rcm_cuda(d_row_ptr, d_col_idx, n, total_edges);

    printf("# RCM order:\n");
    for (int i = 0; i < n; ++i) {
        if (i) printf(" ");
        printf("%d", perm[i]);
    }
    printf("\n");

    // Cleanup
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    free(perm);

    return 0;
}